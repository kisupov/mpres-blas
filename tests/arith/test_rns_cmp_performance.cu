#include "hip/hip_runtime.h"
/*
 *  Test for measure the performance of the RNS magnitude comparison algorithms
 *
 *  Copyright 2020 by Konstantin Isupov.
 *
 *  This file is part of the MPRES-BLAS library.
 *
 *  MPRES-BLAS is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  MPRES-BLAS is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with MPRES-BLAS.  If not, see <https://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include "gmp.h"
#include "../../src/rns.cuh"
#include "../logger.cuh"
#include "../timers.cuh"


#define ITERATIONS 1000


/*
 * CUDA tests
 */

__global__ static void testCudaRnsCmp(int * dr, int * dx, int * dy, int vectorSize) {
    for(int i = 0; i < vectorSize; i++){
        dr[i] = cuda::rns_cmp(&dx[i * RNS_MODULI_SIZE], &dy[i * RNS_MODULI_SIZE]);
    }
}

__global__ static void testCudaMrcCmp(int * dr, int * dx, int * dy, int vectorSize) {
    for(int i = 0; i < vectorSize; i++){
        dr[i] = cuda::mrc_compare_rns(&dx[i * RNS_MODULI_SIZE], &dy[i * RNS_MODULI_SIZE]);
    }
}


/*
 * Common methods
 */

static void resetResult(int * r, int vectorSize){
    memset(r, 0, vectorSize * sizeof(int));
}

__global__ static void resetResultCuda(int * r, int vectorSize) {
    for(int i = 0; i < vectorSize; i++){
        r[i] = 0;
    }
}

static void checkResult(int * ref, int * res, int vectorSize){
    int errors = 0;
    for(int i = 0; i < vectorSize; i++){
        if(ref[i] != res[i]){
            errors++;
        }
    }
    if(errors == 0){
        printf("All results match\n");
    }else{
        printf("Count of errors: %i\n", errors);
    }
}

static void fill_random_array(mpz_t *array, int n) {
    mpz_t rndBnd;
    gmp_randstate_t state;
    gmp_randinit_default(state);
    gmp_randseed_ui(state, (unsigned) time(NULL));
    mpz_init(rndBnd);
    mpz_sub_ui(rndBnd, RNS_MODULI_PRODUCT, 1);
    for (int i = 0; i < n; i++) {
        mpz_urandomm(array[i], state, rndBnd);
    }
    gmp_randclear(state);
    mpz_clear(rndBnd);
}


/*
 * Main test
 */

static void run_test(int iterations) {
    InitCpuTimer();
    InitCudaTimer();

    // Multiple-precision host data
    mpz_t * hx  = new mpz_t[iterations];
    mpz_t * hy  = new mpz_t[iterations];
    int   * ref = new int[iterations]; //reference result

    // RNS host data
    int * hrx = new int[iterations * RNS_MODULI_SIZE];
    int * hry = new int[iterations * RNS_MODULI_SIZE];
    int * hres = new int[iterations];

    //GPU data
    int * drx;
    int * dry;
    int * dres;

    //Memory allocation
    for(int i = 0; i < iterations; i++){
        mpz_init(hx[i]);
        mpz_init(hy[i]);
    }

    hipMalloc(&drx, sizeof(int) * RNS_MODULI_SIZE * iterations);
    hipMalloc(&dry, sizeof(int) * RNS_MODULI_SIZE * iterations);
    hipMalloc(&dres, sizeof(int) * iterations);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    //Generate inputs
    fill_random_array(hx, iterations);
    waitFor(5);
    fill_random_array(hy, iterations);

    //Convert to the RNS
    for(int i = 0; i < iterations; i++){
        rns_from_binary(&hrx[i * RNS_MODULI_SIZE], hx[i]);
        rns_from_binary(&hry[i * RNS_MODULI_SIZE], hy[i]);
    }

    // Copying to the GPU
    hipMemcpy(drx, hrx, sizeof(int) * RNS_MODULI_SIZE * iterations, hipMemcpyHostToDevice);
    hipMemcpy(dry, hry, sizeof(int) * RNS_MODULI_SIZE * iterations, hipMemcpyHostToDevice);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();

    // Computing exact results
    //---------------------------------------------------------
    for(int i = 0; i < iterations; i++){
        ref[i] = mpz_cmp(hx[i], hy[i]);
    }
    //---------------------------------------------------------
    Logger::printDash();
    PrintTimerName("[CPU] rns_cmp");
    resetResult(hres, iterations);
    //Launch
    StartCpuTimer();
    for(int i = 0; i < iterations; i++){
        hres[i] = rns_cmp(&hrx[i * RNS_MODULI_SIZE], &hry[i * RNS_MODULI_SIZE]);
    }
    EndCpuTimer();
    PrintCpuTimer("took");
    checkResult(ref, hres, iterations);
    //---------------------------------------------------------
    Logger::printDash();
    PrintTimerName("[CUDA] rns_cmp");
    resetResult(hres, iterations);
    resetResultCuda<<<1,1>>>(dres, iterations);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
    //Launch
    StartCudaTimer();
    testCudaRnsCmp<<<1,1>>>(dres, drx, dry, iterations);
    EndCudaTimer();
    PrintCudaTimer("took");
    //Copying to the host
    hipMemcpy(hres, dres, sizeof(int) * iterations , hipMemcpyDeviceToHost);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
    checkResult(ref, hres, iterations);
    //---------------------------------------------------------
    Logger::printDash();
    PrintTimerName("[CUDA] mrc_compare_rns");
    resetResult(hres, iterations);
    resetResultCuda<<<1,1>>>(dres, iterations);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
    //Launch
    StartCudaTimer();
    testCudaMrcCmp<<<1,1>>>(dres, drx, dry, iterations);
    EndCudaTimer();
    PrintCudaTimer("took");
    //Copying to the host
    hipMemcpy(hres, dres, sizeof(int) * iterations , hipMemcpyDeviceToHost);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
    checkResult(ref, hres, iterations);
    //---------------------------------------------------------
    
    // Cleanup
    for(int i = 0; i < iterations; i++){
        mpz_clear(hx[i]);
        mpz_clear(hy[i]);
    }
    delete [] hx;
    delete [] hy;
    delete [] ref;
    delete [] hrx;
    delete [] hry;
    delete [] hres;
    hipFree(drx);
    hipFree(dry);
    hipFree(dres);
}

int main() {
    hipDeviceReset();
    rns_const_init();
    Logger::beginTestDescription(Logger::RNS_CMP_PERFORMANCE_TEST);
    Logger::printParam("ITERATIONS", ITERATIONS);
    Logger::endSection(true);
    Logger::printSpace();
    run_test(ITERATIONS);
    Logger::endTestDescription();
    return 0;
}