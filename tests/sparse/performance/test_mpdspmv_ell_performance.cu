/*
 *  Performance test for SpMV routines using the ELLPACK matrix format (double precision matrix)
 *  Path to the matrix must be given as a command line argument, e.g., ../../tests/sparse/matrices/t3dl.mtx

 *  Copyright 2020 by Konstantin Isupov and Ivan Babeshko.
 *
 *  This file is part of the MPRES-BLAS library.
 *
 *  MPRES-BLAS is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  MPRES-BLAS is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with MPRES-BLAS.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "logger.cuh"
#include "tsthelper.cuh"
#include "sparse/matrix_converter.cuh"
#include "sparse/performance/ellpack/test_mpres_mpdspmv_ellpack_scalar.cuh"
#include "sparse/performance/ellpack/test_campary_mpdspmv_ellpack.cuh"
#include "sparse/performance/ellpack/test_cump_mpspmv_ellpack.cuh"
#include "sparse/performance/ellpack/test_double_spmv_ellpack.cuh"
#include "sparse/performance/csr/test_taco_spmv_csr.cuh"

int INP_BITS; //in bits
int INP_DIGITS; //in decimal digits

void setPrecisions() {
    INP_BITS = (int) (MP_PRECISION / 4);
    INP_DIGITS = (int) (INP_BITS / 3.32 + 1);
}

void initialize() {
    hipDeviceReset();
    rns_const_init();
    mp_const_init();
    setPrecisions();
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
}

void finalize() {
}

void test(const char * MATRIX_PATH, const int M, const int N, const int LINES, const int NZR, const bool SYMM, const string DATATYPE) {
    //Input arrays
    mpfr_t *vectorX = create_random_array(N, INP_BITS);
    auto *AS = new double [M * NZR]();
    auto *JA = new int[M * NZR]();
    //Convert a sparse matrix to the double-precision ELLPACK format
    convert_to_ellpack(MATRIX_PATH, M, NZR, LINES, SYMM, AS, JA);
    //Launch tests
    test_double_spmv_ellpack(M, N, NZR, JA, AS, vectorX);
    //test_taco_spmv_csr(MATRIX_PATH, vectorX, DATATYPE);
    test_mpres_mpdspmv_ellpack_scalar(M, N, NZR, JA, AS, vectorX);
    test_campary_mpdspmv_ellpack<CAMPARY_PRECISION>(M, N, NZR, JA, AS, vectorX, INP_DIGITS);
    test_cump_mpspmv_ellpack(M, N, NZR, JA, AS, vectorX, MP_PRECISION, INP_DIGITS);
    checkDeviceHasErrors(hipDeviceSynchronize());
    // cudaCheckErrors(); //CUMP gives failure
    //Cleanup
    for(int i = 0; i < N; i++){
        mpfr_clear(vectorX[i]);
    }
    delete[] vectorX;
    delete[] AS;
    delete[] JA;
    hipDeviceReset();
}

int main(int argc, char *argv[]) {

    //The operation parameters. Read from an input file that contains a sparse matrix
    int M = 0; //number of rows
    int N = 0; //number of columns
    int NZR = 0; //number of nonzeros per row array (maximum number of nonzeros per row in the matrix A)
    int NZMD = 0; //number of nonzeros in the main diagonal of the matrix
    int LINES = 0; //number of lines in the input matrix file
    bool SYMM = false; //true if the input matrix is to be treated as symmetrical; otherwise false
    string DATATYPE; //defines type of data in MatrixMarket: real, integer, binary

    initialize();

    //Start logging
    Logger::beginTestDescription(Logger::SPMV_MPD_ELLPACK_PERFORMANCE_TEST);
    if(argc<=1) {
        printf("Matrix is not specified in command line arguments.");
        Logger::printSpace();
        Logger::endTestDescription();
        exit(1);
    }
    const char * MATRIX_PATH = argv[1];

    Logger::beginSection("Operation info:");
    Logger::printParam("Matrix path", MATRIX_PATH);
    read_matrix_properties(MATRIX_PATH, M, N, LINES, NZR, NZMD, SYMM, DATATYPE);
    Logger::printParam("Number of rows in matrix, M", M);
    Logger::printParam("Number of column in matrix, N", N);
    Logger::printParam("Number of nonzeros in matrix, NNZ", SYMM ? ( (LINES - NZMD) * 2 + NZMD) : LINES);
    Logger::printParam("Number of nonzeros per row array, NZR", NZR);
    Logger::printParam("Symmetry of matrix, SYMM", SYMM);
    Logger::printParam("Data type, DATATYPE", DATATYPE);
    Logger::printDash();
    Logger::beginSection("Additional info:");
    Logger::printParam("RNS_MODULI_SIZE", RNS_MODULI_SIZE);
    Logger::printParam("MP_PRECISION", MP_PRECISION);
    Logger::printParam("CAMPARY_PRECISION (n-double)", CAMPARY_PRECISION);
    Logger::endSection(true);

    //Run the test
    test(MATRIX_PATH, M, N, LINES, NZR, SYMM, DATATYPE);

    //Finalize
    finalize();

    //End logging
    Logger::endTestDescription();

    return 0;
}