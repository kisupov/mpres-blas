/*
 *  Performance test for SpMV routines using the JAD (JDS) matrix format (multiple precision matrix)
 *  Path to the matrix must be given as a command line argument, e.g., ../../tests/sparse/matrices/t3dl.mtx

 *  Copyright 2020 by Konstantin Isupov and Ivan Babeshko.
 *
 *  This file is part of the MPRES-BLAS library.
 *
 *  MPRES-BLAS is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  MPRES-BLAS is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with MPRES-BLAS.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "logger.cuh"
#include "tsthelper.cuh"
#include "sparse/matrix_converter.cuh"
#include "sparse/performance/jad/test_mpres_mpspmv_jad_scalar.cuh"
#include "sparse/performance/jad/test_campary_mpspmv_jad.cuh"
#include "sparse/performance/jad/test_cump_mpspmv_jad.cuh"
#include "sparse/performance/jad/test_double_spmv_jad.cuh"
#include "sparse/performance/csr/test_taco_spmv_csr.cuh"

int INP_BITS; //in bits
int INP_DIGITS; //in decimal digits

void setPrecisions() {
    INP_BITS = (int) (MP_PRECISION / 4);
    INP_DIGITS = (int) (INP_BITS / 3.32 + 1);
}

void initialize() {
    hipDeviceReset();
    rns_const_init();
    mp_const_init();
    setPrecisions();
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
}

void finalize() {
}

void test(const char * MATRIX_PATH, const int M, const int N, const int LINES, const int NZR, const int NNZ, const bool SYMM, const string DATATYPE) {

    //Input arrays
    mpfr_t *vectorX = create_random_array(N, INP_BITS);
    auto *AS = new double [NNZ]();
    auto *JA = new int[NNZ]();
    auto *JCP = new int[NZR + 1]();
    auto *PERM_ROWS = new int[M]();

    //Convert a sparse matrix to the double-precision JAD (JDS) format
    convert_to_jad(MATRIX_PATH, M, NZR, NNZ, LINES, SYMM, AS, JCP, JA, PERM_ROWS);

    //Launch tests
    test_double_spmv_jad(M, N, NZR, NNZ, JA, JCP, AS, PERM_ROWS, vectorX);
    //test_taco_spmv_csr(MATRIX_PATH, vectorX, DATATYPE);
    test_mpres_mpspmv_jad(M, N, NZR, NNZ, JA, JCP, AS, PERM_ROWS, vectorX);
    test_campary_mpspmv_jad<CAMPARY_PRECISION>(M, N, NZR, NNZ, JA, JCP, AS, PERM_ROWS, vectorX, INP_DIGITS);
    test_cump_mpspmv_jad(M, N, NZR, NNZ, JA, JCP, AS, PERM_ROWS, vectorX, MP_PRECISION, INP_DIGITS);
    checkDeviceHasErrors(hipDeviceSynchronize());
    // cudaCheckErrors(); //CUMP gives failure

    //Cleanup
    for(int i = 0; i < N; i++){
        mpfr_clear(vectorX[i]);
    }
    delete[] vectorX;
    delete[] AS;
    delete[] JA;
    delete[] JCP;
    delete[] PERM_ROWS;
    hipDeviceReset();
}

int main(int argc, char *argv[]) {

    //The operation parameters. Read from an input file that contains a sparse matrix
    int M = 0; //number of rows
    int N = 0; //number of columns
    int NNZ = 0; //number of nonzeros in matrix
    int NZR = 0; //number of nonzeros per row array (maximum number of nonzeros per row in the matrix A)
    int NZMD = 0; //number of nonzeros in main diagonal of the matrix
    int LINES = 0; //number of lines in the input matrix file
    bool SYMM = false; //true if the input matrix is to be treated as symmetrical; otherwise false
    string DATATYPE; //defines type of data in MatrixMarket: real, integer, binary

    initialize();

    //Start logging
    Logger::beginTestDescription(Logger::SPMV_MP_JAD_PERFORMANCE_TEST);
    if(argc<=1) {
        printf("Matrix is not specified in command line arguments.");
        Logger::printSpace();
        Logger::endTestDescription();
        exit(1);
    }
    const char * MATRIX_PATH = argv[1];

    Logger::beginSection("Operation info:");
    Logger::printParam("Matrix path", MATRIX_PATH);
    read_matrix_properties(MATRIX_PATH, M, N, LINES, NZR, NZMD, SYMM, DATATYPE);
    NNZ = SYMM ? ( (LINES - NZMD) * 2 + NZMD) : LINES;
    Logger::printParam("Number of rows in matrix, M", M);
    Logger::printParam("Number of column in matrix, N", N);
    Logger::printParam("Number of nonzeros in matrix, NNZ", NNZ);
    Logger::printParam("Number of nonzeros per row array, NZR", NZR);
    Logger::printParam("Symmetry of matrix, SYMM", SYMM);
    Logger::printParam("Data type, DATATYPE", DATATYPE);
    Logger::printDash();
    Logger::beginSection("Additional info:");
    Logger::printParam("RNS_MODULI_SIZE", RNS_MODULI_SIZE);
    Logger::printParam("MP_PRECISION", MP_PRECISION);
    Logger::printParam("CAMPARY_PRECISION (n-double)", CAMPARY_PRECISION);
    Logger::endSection(true);

    //Run the test
    test(MATRIX_PATH, M, N, LINES, NZR, NNZ, SYMM, DATATYPE);

    //Finalize
    finalize();

    //End logging
    Logger::endTestDescription();

    return 0;
}