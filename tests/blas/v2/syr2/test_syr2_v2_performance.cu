#include "hip/hip_runtime.h"
/*
 *  Performance test for BLAS SYR2 routines (symmetric rank-2 update)
 *
 *  Copyright 2022 by Konstantin Isupov.
 *
 *  This file is part of the MPRES-BLAS library.
 *
 *  MPRES-BLAS is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  MPRES-BLAS is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with MPRES-BLAS.  If not, see <https://www.gnu.org/licenses/>.
 */

#include "logger.cuh"
#include "tsthelper.cuh"
#include "blas/mblas_enum.cuh"
#include "test_double_syr2.cuh"
#include "test_mpfr_syr2.cuh"
#include "test_openblas_syr2.cuh"
#include "test_cublas_syr2.cuh"
#include "test_mpres_syr2.cuh"
#include "test_campary_syr2.cuh"

#define N 5000  // Number of matrix columns and the vector X dimension
#define LDA (N) // Specifies the leading dimension of A as declared in the calling (sub)program.
#define UPLO mblas_lower // Specifies whether the upper or lower triangular part of the array A is used.
#define INCX 1 // Specifies the increment for the elements of x.
#define INCY 1 // Specifies the increment for the elements of y.
#define REPEAT_TEST 10 //Number of repeats

int MP_PRECISION_DEC;
int INP_BITS;
int INP_DIGITS;

void setPrecisions() {
    MP_PRECISION_DEC = (int) (MP_PRECISION / 3.32 + 1);
    INP_BITS = (int) (MP_PRECISION / 4);
    INP_DIGITS = (int) (INP_BITS / 3.32 + 1);
}

void initialize() {
    hipDeviceReset();
    rns_const_init();
    mp_const_init();
    setPrecisions();
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
}

void finalize() {
    hipDeviceReset();
}

void test() {
    //Actual length of the vectors
    int lenx = (1 + (N - 1) * abs(INCX));
    int leny = (1 + (N - 1) * abs(INCY));

    //Inputs
    mpfr_t *vectorX = create_random_array(lenx, INP_BITS);
    mpfr_t *vectorY = create_random_array(leny, INP_BITS);
    mpfr_t *matrixA = create_random_array(LDA * N, INP_BITS);
    mpfr_t *alpha = create_random_array(1, INP_BITS);

    //Launch tests
    test_openblas(UPLO, N, alpha[0], vectorX, INCX, vectorY, INCY, matrixA, LDA, REPEAT_TEST);
    test_double(UPLO, N, alpha[0], vectorX, INCX, vectorY, INCY, matrixA, LDA, REPEAT_TEST);
    test_mpfr(UPLO, N, alpha[0], vectorX, INCX, vectorY, INCY, matrixA, LDA, REPEAT_TEST);
    test_cublas(UPLO, N, alpha[0], vectorX, INCX, vectorY, INCY, matrixA, LDA, REPEAT_TEST);
    test_double_cuda(UPLO, N, alpha[0], vectorX, INCX, vectorY, INCY, matrixA, LDA, REPEAT_TEST);
    test_mpres_syr2(UPLO, N, alpha[0], vectorX, INCX, vectorY, INCY, matrixA, LDA, REPEAT_TEST);
    test_campary_syr2<CAMPARY_PRECISION>(UPLO, N, alpha[0], vectorX, INCX, vectorY, INCY, matrixA, LDA, INP_DIGITS, REPEAT_TEST);
    checkDeviceHasErrors(hipDeviceSynchronize());
    cudaCheckErrors();
    //Cleanup
    for (int i = 0; i < LDA * N; i++) {
        mpfr_clear(matrixA[i]);
    }
    for (int i = 0; i < lenx; i++) {
        mpfr_clear(vectorX[i]);
    }
    for (int i = 0; i < leny; i++) {
        mpfr_clear(vectorY[i]);
    }
    mpfr_clear(alpha[0]);
    delete[] matrixA;
    delete[] vectorX;
    delete[] vectorY;
    delete[] alpha;
    hipDeviceReset();
}
int main() {
    initialize();
    Logger::beginTestDescription(Logger::BLAS_SYR2_PERFORMANCE_TEST);
    Logger::printTestParameters(N * N, REPEAT_TEST, MP_PRECISION, MP_PRECISION_DEC);
    Logger::beginSection("Operation info:");
    Logger::printParam("Matrix rows and columns, N", N);
    Logger::printParam("LDA", LDA);
    Logger::printParam("INCX", INCX);
    Logger::printParam("INCY", INCY);
    Logger::printParam("UPLO", UPLO);
    Logger::printDash();
    Logger::beginSection("Additional info:");
    Logger::printParam("RNS_MODULI_SIZE", RNS_MODULI_SIZE);
    Logger::printParam("CAMPARY_PRECISION (n-double)", CAMPARY_PRECISION);
    Logger::endSection(true);
    test();
    finalize();
    Logger::endTestDescription();
    return 0;
}